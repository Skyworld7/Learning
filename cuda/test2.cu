#include<stdio.h>
#include "hip/hip_runtime.h"
__global__ void helloFromGPU(){
    printf("Hello World");
}
__global__ void VecAdd(float *A,float *B,float *C){
    int i=threadIdx.x+blockIdx.x*blockDim.x;
    C[i]=A[i]+B[i];
}
int main(){
    
    helloFromGPU<<<1,10>>>();
    hipDeviceReset();
    return 0;
}